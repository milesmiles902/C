#include <sys/types.h>
#include <math.h>
#include <errno.h>

#include <stdio.h>
#include <stdlib.h>

#include <string.h>
#include <jansson.h>
#include <sys/time.h>

#include "point.h"
#include "kmeans.h"
#include "config.h"

int NUMBER_OF_POINTS = 100000;
int NUMBER_OF_CENTROIDS = 10;

void print_me(Centroid* centroids) {

    if (!DEBUG_LOGS) {
        return;
    }

    for (int i = 0; i < NUMBER_OF_CENTROIDS; i++) {
        printf("[x=%lf, y=%lf, x_sum=%lf, y_sum=%lf, num_points=%i]\n", 
               centroids[i].x, centroids[i].y, centroids[i].x_sum,
               centroids[i].y_sum, centroids[i].num_points);
    }

    printf("--------------------------------------------------\n");
}

long int run_kmeans_repo_specifications(Point* points, Centroid* centroids) {
    struct timeval time_before, time_after, time_result;
    gettimeofday(&time_before, NULL);

    // load the initial centroids
    for (int ci = 0; ci < NUMBER_OF_CENTROIDS; ci++) {
        centroids[ci].x = points[ci].x;
        centroids[ci].y = points[ci].y;
    }

    print_me(centroids);

    for (int i = 0; i < TIMES; i++) {

        km_execute(points, centroids, NUMBER_OF_POINTS, NUMBER_OF_CENTROIDS);

        if (i + 1 == TIMES) {
            print_me(centroids);
        } else {
            // load the centroids to next iteration
            for (int ci = 0; ci < NUMBER_OF_CENTROIDS; ci++) {
                centroids[ci].x = points[ci].x;
                centroids[ci].y = points[ci].y;
            }
        }
    }

    gettimeofday(&time_after, NULL);
    timersub(&time_after, &time_before, &time_result);
    long int ms = ((long int)time_result.tv_sec*1000) + ((long int)time_result.tv_usec/1000);

    return ms / TIMES;
}

long int run_kmeans_rocks(Point* points, Centroid* centroids) {
    // load the initial centroids
    for (int i = 0; i < NUMBER_OF_CENTROIDS; i++) {
        centroids[i].x = points[i].x;
        centroids[i].y = points[i].y;
    }

    print_me(centroids);

    struct timeval time_before, time_after, time_result;
    gettimeofday(&time_before, NULL);

    km_execute(points, centroids, NUMBER_OF_POINTS, NUMBER_OF_CENTROIDS);

    gettimeofday(&time_after, NULL);
    timersub(&time_after, &time_before, &time_result);
    long int ms = ((long int)time_result.tv_sec*1000) + ((long int)time_result.tv_usec/1000);

    print_me(centroids);

    return ms; 
}

int main(int argc, char *argv[])
{
    json_t *json;
    json_error_t error;
    size_t index;
    long int total_time = 0;
    json_t *value;

    if (argc > 1 && argc < 4) {
        printf("Usage: ./kmeans.out [input_file.json number_of_points number_of_centroids]\n");
        return 0;
    }

    if (argc == 4) {
        json = json_load_file(argv[1], 0, &error);
        NUMBER_OF_POINTS = atoi(argv[2]);
        NUMBER_OF_CENTROIDS = atoi(argv[3]);
    }
    else {
        json = json_load_file("../points.json", 0, &error);
    }

    hipSetDevice(0);

    // 100.000 points it's the repository default.
    Point* points = (Point*) malloc(NUMBER_OF_POINTS * sizeof(Point));
    Centroid* centroids = (Centroid*) malloc(NUMBER_OF_CENTROIDS * sizeof(Centroid));

    // validates json
    if (!json) {
        printf("Error parsing Json file");
        fflush(stdout);
        return -1;
    }

    // load points from json
    json_array_foreach(json, index, value)
    {
        float x = json_number_value(json_array_get(value, 0));
        float y = json_number_value(json_array_get(value, 1));
        points[index].x = x;
        points[index].y = y;
    }

    // call K-means
    if (REPOSITORY_SPECIFICATION == 1) {
        total_time = run_kmeans_repo_specifications(points, centroids);
    } else {
        total_time = run_kmeans_rocks(points, centroids);
    }

    free(centroids);
    free(points);

    printf("Average Time: %li ms\n", total_time);

    hipDeviceReset();

    return 0;
}
