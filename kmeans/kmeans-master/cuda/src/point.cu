#include "hip/hip_runtime.h"
#include"point.h"

#include <stdlib.h>
#include <stdio.h>

__device__ void km_divide(Point* p, long d) {
    p->x = p->x / ((float) d);
    p->y = p->y / ((float) d);
    return;
}

__device__ void km_add(Point* p1, Point* p2) {
    p1->x = p1->x + p2->x;
    p1->y = p1->y + p2->y;
    return;
}

__device__ void km_sub(Point* p1, Point* p2) {
    p1->x = p1->x - p2->x;
    p1->y = p1->y - p2->y;
    return;
}

__device__ float km_sq(float x) {
    return x * x;
}

__device__ float km_modulus(Point* p) {
    return sqrtf(km_sq(p->x) + km_sq(p->y));
}

__device__ float km_distance(Point* p, Centroid* c)
{
    //printf("valor %lf\n", &c->x);
    float dx = p->x - c->x;
    float dy = p->y - c->y;
    return sqrtf(dx*dx + dy*dy);
}
