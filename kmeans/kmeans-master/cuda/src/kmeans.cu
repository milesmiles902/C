#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"
#include "point.h"
#include "config.h"

/**
    Groups the points in a centroid.
*/
__global__ void km_group_by_cluster(Point* points, Centroid* centroids,
        int num_centroids, int num_points)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    int i = 0;

    float minor_distance = -1.0;

	if (idx < num_points) {
	    for (i = 0; i < num_centroids; i++) {
	        float my_distance = km_distance(&points[idx], &centroids[i]);
	
	        // if my_distance is less than the lower minor_distance 
	        // or minor_distance is not yet started
	        if (minor_distance > my_distance || minor_distance == -1.0) {
	            minor_distance = my_distance;
	            points[idx].cluster = i;
	        }
	    }
	}
}

/**
    Sum the points of each centroid
*/
__global__ void km_sum_points_cluster(Point* points, Centroid* centroids,
        int num_centroids, int num_points)
{
    extern __shared__ Centroid s_centroids[];
    
    int tdx = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // init global memory
    if (idx < num_centroids) {
        centroids[idx].x_sum = 0.0;
        centroids[idx].y_sum = 0.0;
        centroids[idx].num_points = 0.0;
    }

    // init shared memory
    if (tdx < num_centroids) {
        s_centroids[tdx].x_sum = 0.0;
        s_centroids[tdx].y_sum = 0.0;
        s_centroids[tdx].num_points = 0.0;
    }

    __syncthreads();
    
    // use shared memory for intermediate sums
    if (idx < num_points) {
        int i = points[idx].cluster;
        atomicAdd(&s_centroids[i].x_sum, points[idx].x);
        atomicAdd(&s_centroids[i].y_sum, points[idx].y);
        atomicAdd(&s_centroids[i].num_points, 1);
	}

    __syncthreads();
    
    // then sum partial results in global memory, thus we reduce accesses to global memory
    if (tdx < num_centroids) {
        atomicAdd(&centroids[tdx].x_sum, s_centroids[tdx].x_sum);
        atomicAdd(&centroids[tdx].y_sum, s_centroids[tdx].y_sum);
        atomicAdd(&centroids[tdx].num_points, s_centroids[tdx].num_points);
    }
}

/**
    Update the centroids with current clustering.
    Gets the x and y sum and divides by number of point for each centroid.\
*/
__global__ void km_update_centroids(Centroid* centroids, int num_centroids)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < num_centroids) {
	    if (centroids[idx].num_points > 0) {
	        centroids[idx].x = centroids[idx].x_sum / centroids[idx].num_points;
	        centroids[idx].y = centroids[idx].y_sum / centroids[idx].num_points;
	    }
	}
}

/**
    Compare the clusters of each point.
    @param p1 - points of current iteration
    @param p2 - points of last iteration
*/
__global__ void km_points_compare(Point* p1, Point* p2, int num_points,
        int *result)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_points) {
        // if any points has its cluster different, changes the result variable
        if (p1[idx].cluster != p2[idx].cluster) {
            *result = 0;
        }
    }
}

/**
    Copy a point array.
    Utilized to copy the status of points on the last iteration to compare them.
*/
__global__ void km_points_copy(Point* p_dest, Point* p_src, int num_points)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_points) {
        p_dest[idx] = p_src[idx];
    }
}

/**
* Executes the k-mean algorithm.
* To measure your global methods, use that:
*
*    hipEvent_t start, stop;
*    float time;
*    hipEventCreate(&start);
*    hipEventCreate(&stop);
*    hipEventRecord(start, 0);
*
* //  put your__global__ method here!
*
*    hipEventRecord(stop, 0);
*    hipEventSynchronize(stop);
*    hipEventElapsedTime(&time, start, stop);
*    printf("%lf\n", times)
*/
void km_execute(Point* h_points, Centroid* h_centroids, int num_points,
        int num_centroids)
{
    int iterations = 0;
    Point* d_points;
    Point* d_points_old;
    Centroid* d_centroids;
    int h_res = 1;
    int *d_res;

    hipMalloc((void**) &d_res, sizeof(int));
    hipMalloc((void**) &d_points_old, sizeof(Point) * num_points);
    hipMalloc((void **) &d_points, sizeof(Point) * num_points);
    hipMalloc((void **) &d_centroids, sizeof(Centroid) * num_centroids);

    hipMemcpy(d_points, h_points, sizeof(Point) * num_points, hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, h_centroids, sizeof(Centroid) * num_centroids, hipMemcpyHostToDevice);   

    while (true) {

        km_group_by_cluster<<<ceil(num_points/100), 100>>>(d_points, d_centroids,
                num_centroids, num_points);
        hipDeviceSynchronize();
        
        km_sum_points_cluster<<<ceil(num_points/100), 100, num_centroids*sizeof(Centroid)>>>(d_points, d_centroids,
                num_centroids, num_points);
        hipDeviceSynchronize();

        km_update_centroids<<<ceil(num_centroids/10), 10>>>(d_centroids, num_centroids);
        hipDeviceSynchronize();

        if (REPOSITORY_SPECIFICATION == 1) {
            // in repository specifications, 
            // we just want know if number of 
            // iterations is equals NUMBER_OF_ITERATIONS - 1 (iterations starts in 0)
            if (iterations == (NUMBER_OF_ITERATIONS - 1)) {
                break;
            }
        } else {
            // TODO: WARNING:
            // THIS IMPLEMENTATION IS NOT WORKING YET!
            if (iterations > 0) {
                h_res = 1;
                hipMemcpy(d_res, &h_res , sizeof(int), hipMemcpyHostToDevice);
                km_points_compare<<<ceil(num_points/10), 10>>>(d_points, d_points_old,
                        num_points, d_res);
                hipDeviceSynchronize();

                hipMemcpy(&h_res, d_res, sizeof(int), hipMemcpyDeviceToHost);

                // if h_rest == 1 the two vector of points are equal and the kmeans iterations
                // has completed all work
                if (h_res == 1) {
                    break;
                }
            }

            km_points_copy<<<ceil(num_points/100), 100>>>(d_points_old, d_points,
                num_points);
            hipDeviceSynchronize();
        }
        
        iterations++;
    }

    hipMemcpy(h_centroids, d_centroids , sizeof(Centroid) * num_centroids, hipMemcpyDeviceToHost);

    hipFree(d_points);
    hipFree(d_centroids);
    hipFree(d_points_old);
    hipFree(d_res);
}
